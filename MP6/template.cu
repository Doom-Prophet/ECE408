#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

typedef unsigned char uint8_t;
typedef unsigned int  uint_t;

#define HISTOGRAM_LENGTH 256
#define NUM_CHANNELS 3

//@@ insert code here
//@@ Cast the image from float to unsigned char @@//
__global__ void float2Uint8(float *input, uint8_t *output, int width, int height){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
      int idx = blockIdx.z * width * height + y * width + x;
      output[idx] = (uint8_t)(255 * input[idx]);
    }
}

//@@ Convert the image from RGB to GrayScale @@//
__global__ void rgb2GrayScale(uint8_t *input, uint8_t *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * (width) + x;
        uint8_t r = input[3 * idx + 0];
        uint8_t g = input[3 * idx + 1];
        uint8_t b = input[3 * idx + 2];
        output[idx] = (uint8_t) (0.21*r + 0.71*g + 0.07*b);
    }
}

//@@ Compute the histogram of grayImage @@//
__global__ void grayScale2Hist(uint8_t *input, uint_t *output, int width, int height) {

    __shared__ uint_t histogram[HISTOGRAM_LENGTH];

    int tIdx = threadIdx.x + threadIdx.y * blockDim.x;
    if (tIdx < HISTOGRAM_LENGTH) {
        histogram[tIdx] = 0;
    }

    __syncthreads();
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * (width) + x;
        uint8_t val = input[idx];
        atomicAdd(&(histogram[val]), 1);
    }

    __syncthreads();
    if (tIdx < HISTOGRAM_LENGTH) {
        atomicAdd(&(output[tIdx]), histogram[tIdx]);
    }
}

//@@ Compute the Cumulative Distribution Function of histogram @@//
__global__ void histogram2CDF(uint_t *input, float *output, int width, int height) {
    __shared__ uint_t cdf[HISTOGRAM_LENGTH];
    int x = threadIdx.x;
    cdf[x] = input[x];

    //** Parallel inclusive scan algorithm (based on Brent-Kung) **//
    // First scan half
    for (unsigned int stride = 1; stride <= HISTOGRAM_LENGTH / 2; stride *= 2) {
        __syncthreads();
        int idx = (x + 1) * 2 * stride - 1;
        if (idx < HISTOGRAM_LENGTH) {
            cdf[idx] += cdf[idx - stride];
        }
    }

    // Second scan half
    for (int stride = HISTOGRAM_LENGTH / 4; stride > 0; stride /= 2) {
        __syncthreads();
        int idx = (x + 1) * 2 * stride - 1;
        if (idx + stride < HISTOGRAM_LENGTH) {
            cdf[idx + stride] += cdf[idx];
        }
    }

    __syncthreads();
    output[x] = cdf[x] / ((float) (width * height));
}

//@@ Compute the minimum value of the CDF @@//
//@@ 1. Define the histogram equalization function and apply the histogram equalization function @@//
__global__ void equalizeImage(uint8_t *inout, float *cdf, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = blockIdx.z * (width * height) + y * (width) + x;
        uint8_t val = inout[idx];

        float equalized = 255 * (cdf[val] - cdf[0]) / (1.0 - cdf[0]);
        float clamped   = min(max(equalized, 0.0), 255.0);

        inout[idx] = (uint8_t) (clamped);
    }
}

//@@ 2. Cast back to float @@//
__global__ void uInt82Float(uint8_t *input, float *output, int width, int height) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = blockIdx.z * (width * height) + y * (width) + x;
        output[idx] = (float) (input[idx] / 255.0);
    }
}

int main(int argc, char **argv) {
    wbArg_t args;

    int imageWidth;
    int imageHeight;
    int imageChannels;

    wbImage_t inputImage;
    wbImage_t outputImage;

    float *hostInputImageData;
    float *hostOutputImageData;
    const char *inputImageFile;

    //@@ Insert more code here
    float   *deviceImageFloat;
    uint8_t *deviceImageUChar;
    uint8_t *deviceImageUCharGrayScale;
    uint_t  *deviceImageHistogram;
    float   *deviceImageCDF;

    args = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(args, 0);

    wbTime_start(Generic, "Importing data and creating memory on host");
    inputImage = wbImport(inputImageFile);
    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);
    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
    hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    //@@ insert code here
    //*** Allocating GPU memory ***//
    hipMalloc((void**) &deviceImageFloat, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void**) &deviceImageUChar, imageWidth * imageHeight * imageChannels * sizeof(uint8_t));
    hipMalloc((void**) &deviceImageUCharGrayScale, imageWidth * imageHeight * sizeof(uint8_t));
    hipMalloc((void**) &deviceImageHistogram, HISTOGRAM_LENGTH * sizeof(uint_t));
    hipMemset((void *) deviceImageHistogram, 0, HISTOGRAM_LENGTH * sizeof(uint_t));
    hipMalloc((void**) &deviceImageCDF, HISTOGRAM_LENGTH * sizeof(float));

    //*** Copying input memory to the GPU ***//
    hipMemcpy(deviceImageFloat, hostInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyHostToDevice);

    //*** Performing CUDA computation ***//
    dim3 dimGrid;
    dim3 dimBlock;

    //to uint8
    dimGrid  = dim3(ceil(imageWidth/32.0), ceil(imageHeight/32.0), imageChannels);
    dimBlock = dim3(32, 32, 1);

    float2Uint8<<<dimGrid, dimBlock>>>(deviceImageFloat, deviceImageUChar, imageWidth, imageHeight);
    hipDeviceSynchronize();


    //to gray
    dimGrid  = dim3(ceil(imageWidth/32.0), ceil(imageHeight/32.0), 1);
    dimBlock = dim3(32, 32, 1);

    rgb2GrayScale<<<dimGrid, dimBlock>>>(deviceImageUChar, deviceImageUCharGrayScale, imageWidth, imageHeight);
    hipDeviceSynchronize();


    //to histo
    dimGrid  = dim3(ceil(imageWidth/32.0), ceil(imageHeight/32.0), 1);
    dimBlock = dim3(32, 32, 1);

    grayScale2Hist<<<dimGrid, dimBlock>>>(deviceImageUCharGrayScale, deviceImageHistogram, imageWidth, imageHeight);
    hipDeviceSynchronize();


    //to cdf
    dimGrid  = dim3(1, 1, 1);
    dimBlock = dim3(HISTOGRAM_LENGTH, 1, 1);

    histogram2CDF<<<dimGrid, dimBlock>>>(deviceImageHistogram, deviceImageCDF, imageWidth, imageHeight);
    hipDeviceSynchronize();


    //equalize
    dimGrid  = dim3(ceil(imageWidth/32.0), ceil(imageHeight/32.0), imageChannels);
    dimBlock = dim3(32, 32, 1); 

    equalizeImage<<<dimGrid, dimBlock>>>(deviceImageUChar, deviceImageCDF, imageWidth, imageHeight
    );
    hipDeviceSynchronize();


    //to uint8
    dimGrid  = dim3(ceil(imageWidth/32.0), ceil(imageHeight/32.0), imageChannels);
    dimBlock = dim3(32, 32, 1);

    uInt82Float<<<dimGrid, dimBlock>>>(deviceImageUChar, deviceImageFloat, imageWidth, imageHeight);
    hipDeviceSynchronize();

    //*** Copying output memory to the CPU ***//
    hipMemcpy(hostOutputImageData, deviceImageFloat, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceImageFloat);
    hipFree(deviceImageUChar);
    hipFree(deviceImageUCharGrayScale);
    hipFree(deviceImageHistogram);
    hipFree(deviceImageCDF);

    wbSolution(args, outputImage);

    //@@ insert code here
    free(inputImage);
    free(outputImage);
    
    return 0;
}
